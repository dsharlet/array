// Workarounds so that we don't need to install the CUDA toolkit.
// This is just a build test.
#if defined(__CUDA__)
#ifndef __device__
#define __device__ __attribute__((device))
#endif

#ifndef __host__
#define __host__ __attribute__((host))
#endif

// For some reason, this doesn't compile when it's size_t.
__device__ void* malloc(int size);
__device__ void free(void* p);
template<typename... Args>
__device__ int printf(const char* format, Args...);
#endif  // defined(__CUDA__)

#include "array/array.h"

namespace nda {

NDARRAY_HOST_DEVICE
void array_ref_indices() {
  int data[100];
  for (int i = 0; i < 100; i++) {
    data[i] = i;
  }
  dense_array_ref<int, 1> ref_1d(data, {100});
}

NDARRAY_HOST_DEVICE
void reinterpret() {
  float eight = 8.0f;
  int eight_int = *reinterpret_cast<int*>(&eight);

  dense_array_ref<int, 3> int_array(&eight_int, {1, 1, 1});
  dense_array_ref<float, 3> float_array = reinterpret<float>(int_array);
  (void)int_array;
  (void)float_array;
}

NDARRAY_HOST_DEVICE
void array_ref_empty() {
  // This does *not* work: it that shape_ = new_shape is not allowed because we use the
  // defaulted assignment operator, which is apparently __host__ only? This seems
  // like a bug in clang, the operator is explicitly defaulted with a __device__ annotation:
  //   NDARRAY_HOST_DEVICE
  //   shape& operator=(const shape&) = default;
  //
  // dense_array_ref<int, 1> null_ref(nullptr, {10});
  // null_ref.set_shape({{3, 3}}, 3);

  int x;
  array_ref_of_rank<int, 0> scalar_ref(&x, {});
  array_ref_of_rank<int, 0> null_scalar_ref(nullptr, {});
}

// TODO(jiawen): Add CUDA support to image.h, matrix.h, and einsum.h.

}  // namespace nda
